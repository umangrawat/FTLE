#include "hip/hip_runtime.h"
#include "cudaIntegrator.h"
#include "linalgCUDA.h"

/*
MAKEDEVICE int getIndexInputGrid(int x , int y , int z, int* dim)
{
    /// return index, 3 times since values
    int index = (z*dim[1]*dim[0]+ (y)*dim[0]+x)*3;
    return index;
}
*/

////changed 3D to 2D
MAKEDEVICE int getIndexInputGrid(int x , int y, int* dim)
{
    /// return index, 2 times since values
    int index = ((y)*dim[0]+x)*2;
    return index;
}

/*
MAKEDEVICE void getDataPoint(vec3 dataVec, int xPos, int yPos, int zPos,
                             float* inputGrid, int* dimInputGrid)
{
    int index = getIndexInputGrid(xPos,yPos,zPos,dimInputGrid);
    dataVec[0] = inputGrid[index+0];
    dataVec[1] = inputGrid[index+1];
    dataVec[2] = inputGrid[index+2];
}
*/

////changed 3 to 2
MAKEDEVICE void getDataPoint(vec2 dataVec, int xPos, int yPos,
                             float* inputGrid, int* dimInputGrid)
{
    int index = getIndexInputGrid(xPos,yPos,dimInputGrid);
    dataVec[0] = inputGrid[index+0];
    dataVec[1] = inputGrid[index+1];
    ////dataVec[2] = inputGrid[index+2];
}

/*
MAKEDEVICE bool interpolate(vec3 location, vec3 dataVec, float *inputGrid, double *origin,
            int *dimInputGrid, double *spacingInputGrid, int intDirection) {
    vec3 loc;
    d_vec3copy(location,loc);
    //
    //loc[0] -= origin[0];
    //loc[1] -= origin[1];
    //loc[2] -= origin[2];
    //TODO Check local index calculation
    int xPos = (int) (loc[0] / spacingInputGrid[0]);
    int yPos = (int) (loc[1] / spacingInputGrid[1]);
    int zPos = (int) (loc[2] / spacingInputGrid[2]);

    double xLocal = loc[0] - double(xPos)*spacingInputGrid[0];
    double yLocal = loc[1] - double(yPos)*spacingInputGrid[1];
    double zLocal = loc[2] - double(zPos)*spacingInputGrid[2];
    xLocal /= spacingInputGrid[0];
    yLocal /= spacingInputGrid[1];
    zLocal /= spacingInputGrid[2];


    if(xPos > dimInputGrid[0] || yPos > dimInputGrid[1]
       || zPos > dimInputGrid[2]  || xPos < 0 || yPos < 0 || zPos <0) {
        return false;
    }


    vec3 a,b,c,d,e,f,g,h;
    getDataPoint(a,xPos,yPos,zPos,inputGrid, dimInputGrid);
    getDataPoint(b,xPos+1,yPos,zPos,inputGrid, dimInputGrid);
    getDataPoint(c,xPos,yPos+1,zPos,inputGrid, dimInputGrid);
    getDataPoint(d,xPos+1,yPos+1,zPos,inputGrid, dimInputGrid);
    getDataPoint(e,xPos,yPos,zPos+1,inputGrid, dimInputGrid);
    getDataPoint(f,xPos+1,yPos,zPos+1,inputGrid, dimInputGrid);
    getDataPoint(g,xPos,yPos+1,zPos+1,inputGrid, dimInputGrid);
    getDataPoint(h,xPos+1,yPos+1,zPos+1,inputGrid, dimInputGrid);


    d_vec3trilint(a,b,c,d,e,f,g,h, xLocal, yLocal, zLocal, dataVec);
    if(intDirection == -1)
        d_vec3scal(dataVec, -1., dataVec);
    return true;
}
*/

////changed 3 to 2
MAKEDEVICE bool interpolate(vec2 location, vec2 dataVec, float *inputGrid, double *origin,
            int *dimInputGrid, double *spacingInputGrid, int intDirection) {
    vec2 loc;
    d_vec2copy(location,loc);
    //
    //loc[0] -= origin[0];
    //loc[1] -= origin[1];
    //loc[2] -= origin[2];
    //TODO Check local index calculation
    int xPos = (int) (loc[0] / spacingInputGrid[0]);
    int yPos = (int) (loc[1] / spacingInputGrid[1]);
    ////int zPos = (int) (loc[2] / spacingInputGrid[2]);

    double xLocal = loc[0] - double(xPos)*spacingInputGrid[0];
    double yLocal = loc[1] - double(yPos)*spacingInputGrid[1];
    ////double zLocal = loc[2] - double(zPos)*spacingInputGrid[2];
    xLocal /= spacingInputGrid[0];
    yLocal /= spacingInputGrid[1];
    ////zLocal /= spacingInputGrid[2];


    if(xPos > dimInputGrid[0] || yPos > dimInputGrid[1]
       || xPos < 0 || yPos < 0) {
        return false;
    }

////removed e,f,g,h
    vec2 a,b,c,d;
    getDataPoint(a,xPos,yPos,inputGrid, dimInputGrid);
    getDataPoint(b,xPos+1,yPos,inputGrid, dimInputGrid);
    getDataPoint(c,xPos,yPos+1,inputGrid, dimInputGrid);
    getDataPoint(d,xPos+1,yPos+1,inputGrid, dimInputGrid);
    ////getDataPoint(e,xPos,yPos,zPos+1,inputGrid, dimInputGrid);
    ////getDataPoint(f,xPos+1,yPos,zPos+1,inputGrid, dimInputGrid);
    ////getDataPoint(g,xPos,yPos+1,zPos+1,inputGrid, dimInputGrid);
    ////getDataPoint(h,xPos+1,yPos+1,zPos+1,inputGrid, dimInputGrid);

    ////changed trilint to bilint
    d_vec2bilint(a,b,c,d, xLocal, yLocal, dataVec);
    if(intDirection == -1)
        d_vec2scal(dataVec, -1., dataVec);
    return true;
}

/*
MAKEDEVICE double integratePoint(vec3 &location, float *inputGrid, double *origin, int *dimInputGrid, double *spacingInputGrid,
               int integrationDirection, double stepSize, double &stagnation) {


    double intTime = 1;
    vec3 dataVec;
    d_vec3set(dataVec,0,0,0);
    stagnation = 0.;
    ///


    if(false) {

        if (!interpolate(location, dataVec, inputGrid, origin, dimInputGrid, spacingInputGrid, integrationDirection)) {
            stagnation = 0.;
            return intTime;
        }
        ///Simple Euler step

        intTime = stepSize / d_vec3mag(dataVec);

        stagnation = d_vec3mag(dataVec);


        d_vec3nrm(dataVec, dataVec);
        d_vec3scal(dataVec, stepSize, dataVec);
        //d_vec3add(location, dataVec, location);
        d_vec3copy(location,dataVec);
        stagnation = 0.;
        return intTime;
    }
    ///
    /// Runge Kutta integration scheme






    double currentIntTime = 0;
    double currentIntLength = 0;
    vec3 currentLocation;
    vec3 vecNextLocation;
    vec3 currentLocationSave;
    vec3 k1,k2,k3,k4;
    vec3 k1norm, k2norm, k3norm, k4norm;
    vec3 k1scal, k2scal, k3scal, k4scal;
    vec3 step;
    vec3 temp;
    d_vec3copy(location,currentLocation);
    double interStepWeights[3] = {.5,.5,1.0};
    double finalStepWeights[4] = {1.0/6.0, 1.0/3.0,1.0/3.0,1.0/6.0};


    d_vec3copy(currentLocation, currentLocationSave);
    if(!interpolate(currentLocation, vecNextLocation, inputGrid, origin, dimInputGrid, spacingInputGrid, integrationDirection))
    {

        stagnation = 0.;
        return intTime;
    }

    d_vec3copy(vecNextLocation,k1);
    d_vec3nrm(k1,k1norm);
    d_vec3scal(k1, stepSize,k1scal);

    d_vec3copy(k1scal,temp);
    d_vec3scal(temp, interStepWeights[0],temp);
    d_vec3copy(currentLocationSave, currentLocation);
    d_vec3add(currentLocation,temp,currentLocation);

    if(!interpolate(currentLocation, vecNextLocation, inputGrid, origin, dimInputGrid, spacingInputGrid, integrationDirection))
    {

        stagnation = 0.;
        return intTime;
    }
    d_vec3copy(vecNextLocation,k2);
    d_vec3nrm(k2,k2norm);
    d_vec3scal(k2,stepSize,k2scal);

    d_vec3copy(k2scal,temp);
    d_vec3scal(temp,interStepWeights[1],temp);
    d_vec3copy(currentLocationSave, currentLocation);
    d_vec3add(currentLocation,temp,currentLocation);

    if(!interpolate(currentLocation, vecNextLocation, inputGrid, origin, dimInputGrid, spacingInputGrid, integrationDirection))
    {

        stagnation = 0.;
        return intTime;
    }

    d_vec3copy(vecNextLocation,k3);
    d_vec3nrm(k3,k3norm);
    d_vec3scal(k3,stepSize,k3scal);

    d_vec3copy(k3scal,temp);
    d_vec3scal(temp,interStepWeights[2],temp);
    d_vec3copy(currentLocationSave, currentLocation);
    d_vec3add(currentLocation,temp,currentLocation);

    if(!interpolate(currentLocation, vecNextLocation, inputGrid, origin, dimInputGrid, spacingInputGrid, integrationDirection))
    {

        stagnation = 0.;
        return intTime;
    }

    d_vec3copy(vecNextLocation,k4);
    d_vec3nrm(k4,k4norm);
    d_vec3scal(k4,stepSize,k4scal);

    d_vec3scal(k1norm, finalStepWeights[0],temp);
    d_vec3copy(temp, step);
    d_vec3scal(k2norm, finalStepWeights[1], temp);
    d_vec3add(step, temp,step);
    d_vec3scal(k3norm, finalStepWeights[2], temp);
    d_vec3add(step, temp,step);
    d_vec3scal(k4norm, finalStepWeights[3],temp);
    d_vec3add(step,temp,step);
    d_vec3nrm(step,step);
    d_vec3scal(step, stepSize, step);
    currentIntLength += d_vec3mag(step);

    vec3 vel;
    d_vec3copy(k1,vel);
    d_vec3scal(k1,finalStepWeights[0],vel);
    d_vec3scal(k2,finalStepWeights[1],temp);
    d_vec3add(vel,temp,vel);
    d_vec3scal(k3,finalStepWeights[2],temp);
    d_vec3add(vel,temp,vel);
    d_vec3scal(k4,finalStepWeights[3],temp);
    d_vec3add(vel,temp,vel);
    currentIntTime += stepSize/d_vec3mag(vel);

    d_vec3copy(currentLocationSave, currentLocation);
    d_vec3add(currentLocation,step,currentLocation);

    stagnation = d_vec3mag(vel);

    ///End of Runge Kutta

    intTime = stepSize/d_vec3mag(vel);


    //stagnation = d_vec3mag(step);


    //d_vec3nrm(vel,vel);
    //d_vec3scal(vel, stepSize, vel);
    d_vec3add(location, step, location);
    return intTime;

}
*/


////changed 3 to 2
MAKEDEVICE double integratePoint(vec2 &location, float *inputGrid, double *origin, int *dimInputGrid, double *spacingInputGrid,
               int integrationDirection, double stepSize, double &stagnation) {


    double intTime = 1;
    vec2 dataVec;
    d_vec2set(dataVec,0,0);
    stagnation = 0.;
    ///


    if(false) {

        if (!interpolate(location, dataVec, inputGrid, origin, dimInputGrid, spacingInputGrid, integrationDirection)) {
            stagnation = 0.;
            return intTime;
        }
        ///Simple Euler step

        intTime = stepSize / d_vec2mag(dataVec);

        stagnation = d_vec2mag(dataVec);


        d_vec2nrm(dataVec, dataVec);
        d_vec2scal(dataVec, stepSize, dataVec);
        //d_vec3add(location, dataVec, location);
        d_vec2copy(location,dataVec);
        stagnation = 0.;
        return intTime;
    }
    ///
    /// Runge Kutta integration scheme


    double currentIntTime = 0;
    double currentIntLength = 0;
    vec2 currentLocation;
    vec2 vecNextLocation;
    vec2 currentLocationSave;
    vec2 k1,k2,k3,k4;
    vec2 k1norm, k2norm, k3norm, k4norm;
    vec2 k1scal, k2scal, k3scal, k4scal;
    vec2 step;
    vec2 temp;
    d_vec2copy(location,currentLocation);
    double interStepWeights[3] = {.5,.5,1.0};
    double finalStepWeights[4] = {1.0/6.0, 1.0/3.0,1.0/3.0,1.0/6.0};


    d_vec2copy(currentLocation, currentLocationSave);
    if(!interpolate(currentLocation, vecNextLocation, inputGrid, origin, dimInputGrid, spacingInputGrid, integrationDirection))
    {

        stagnation = 0.;
        return intTime;
    }

    d_vec2copy(vecNextLocation,k1);
    d_vec2nrm(k1,k1norm);
    d_vec2scal(k1, stepSize,k1scal);

    d_vec2copy(k1scal,temp);
    d_vec2scal(temp, interStepWeights[0],temp);
    d_vec2copy(currentLocationSave, currentLocation);
    d_vec2add(currentLocation,temp,currentLocation);

    if(!interpolate(currentLocation, vecNextLocation, inputGrid, origin, dimInputGrid, spacingInputGrid, integrationDirection))
    {

        stagnation = 0.;
        return intTime;
    }
    d_vec2copy(vecNextLocation,k2);
    d_vec2nrm(k2,k2norm);
    d_vec2scal(k2,stepSize,k2scal);

    d_vec2copy(k2scal,temp);
    d_vec2scal(temp,interStepWeights[1],temp);
    d_vec2copy(currentLocationSave, currentLocation);
    d_vec2add(currentLocation,temp,currentLocation);

    if(!interpolate(currentLocation, vecNextLocation, inputGrid, origin, dimInputGrid, spacingInputGrid, integrationDirection))
    {

        stagnation = 0.;
        return intTime;
    }

    d_vec2copy(vecNextLocation,k3);
    d_vec2nrm(k3,k3norm);
    d_vec2scal(k3,stepSize,k3scal);

    d_vec2copy(k3scal,temp);
    d_vec2scal(temp,interStepWeights[2],temp);
    d_vec2copy(currentLocationSave, currentLocation);
    d_vec2add(currentLocation,temp,currentLocation);

    if(!interpolate(currentLocation, vecNextLocation, inputGrid, origin, dimInputGrid, spacingInputGrid, integrationDirection))
    {

        stagnation = 0.;
        return intTime;
    }

    d_vec2copy(vecNextLocation,k4);
    d_vec2nrm(k4,k4norm);
    d_vec2scal(k4,stepSize,k4scal);

    d_vec2scal(k1norm, finalStepWeights[0],temp);
    d_vec2copy(temp, step);
    d_vec2scal(k2norm, finalStepWeights[1], temp);
    d_vec2add(step, temp,step);
    d_vec2scal(k3norm, finalStepWeights[2], temp);
    d_vec2add(step, temp,step);
    d_vec2scal(k4norm, finalStepWeights[3],temp);
    d_vec2add(step,temp,step);
    d_vec2nrm(step,step);
    d_vec2scal(step, stepSize, step);
    currentIntLength += d_vec2mag(step);

    vec2 vel;
    d_vec2copy(k1,vel);
    d_vec2scal(k1,finalStepWeights[0],vel);
    d_vec2scal(k2,finalStepWeights[1],temp);
    d_vec2add(vel,temp,vel);
    d_vec2scal(k3,finalStepWeights[2],temp);
    d_vec2add(vel,temp,vel);
    d_vec2scal(k4,finalStepWeights[3],temp);
    d_vec2add(vel,temp,vel);
    currentIntTime += stepSize/d_vec2mag(vel);

    d_vec2copy(currentLocationSave, currentLocation);
    d_vec2add(currentLocation,step,currentLocation);

    stagnation = d_vec2mag(vel);

    ///End of Runge Kutta

    intTime = stepSize/d_vec2mag(vel);


    //stagnation = d_vec3mag(step);


    //d_vec3nrm(vel,vel);
    //d_vec3scal(vel, stepSize, vel);
    d_vec2add(location, step, location);
    return intTime;

}

/*
#if !defined UNITTEST && !defined CPUEXEC
MAKEDEVICE int getGlobalIdx() {
    int blockId = blockIdx.x
                  + blockIdx.y * gridDim.x
                  + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                   + (threadIdx.z * (blockDim.x * blockDim.y))
                   + (threadIdx.y * blockDim.x)
                   + threadIdx.x;
    return threadId;
}
#endif
*/

////removed z component
#if !defined UNITTEST && !defined CPUEXEC
MAKEDEVICE int getGlobalIdx() {
    int blockId = blockIdx.x
                  + blockIdx.y * gridDim.x;
    int threadId = (threadIdx.y * blockDim.x)
                   + threadIdx.x;
    return threadId;
}
#endif


/// fy refers to the current function value
/// fx and fz to the i-1 and i+1 function value
MAKEDEVICE double d_centralDiff(double fx, double fz, double dist)
{

    return (fz-fx)/(2*dist);
}
MAKEDEVICE double d_forwardDiff(double fy, double fz, double dist)
{
    return (fz-fy)/dist;
}
///Changed  fx and fy because of wrong function call further dowm
MAKEDEVICE double d_backwardDiff(double fx, double fy, double dist)
{
    return (fy-fx)/dist;

}

/*
#if !defined UNITTEST && !defined CPUEXEC
MAKEGLOBAL void computeFTLE(double* ftleField, double* flowMap, int* dimSeedGrid,
                            double* spacingSeedGrid, double intTime, double d_LIMIT_DOUBLE)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;


    int z = index / (dimSeedGrid[0] * dimSeedGrid[1]);
    int y = (index % (dimSeedGrid[0] * dimSeedGrid[1])) / dimSeedGrid[0];
    int x = (index % (dimSeedGrid[0] * dimSeedGrid[1])) % dimSeedGrid[0];
    //if(index < dimSeedGrid[0]*dimSeedGrid[1]*dimSeedGrid[2]) {
     //   int x = blockIdx.x * blockDim.x + threadIdx.x;//(index / (dimSeedGrid[0] * dimSeedGrid[1]));
     //   int y = blockIdx.y * blockDim.y + threadIdx.y;//(index % (dimSeedGrid[0] * dimSeedGrid[1])) / dimSeedGrid[1];
     //   int z = blockIdx.z * blockDim.z + threadIdx.z;//(index % (dimSeedGrid[0] * dimSeedGrid[1])) % dimSeedGrid[1];
    if (x < dimSeedGrid[0] && y < dimSeedGrid[1] && z < dimSeedGrid[2] &&
            x >=0 && y >=0 && z>=0){
        vec3 t1, t2;
        int id;
        vec3 du, dv, dw;


        if (z <= 0) {
            id = getIndexInputGrid(x, y, z, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x, y, z + 1, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            t2[2] = (double)flowMap[id + 2];

            du[2] = d_forwardDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[2] = d_forwardDiff(t1[1], t2[1], spacingSeedGrid[1]);
            dw[2] = d_forwardDiff(t1[2], t2[2], spacingSeedGrid[2]);

        } else if (z >= dimSeedGrid[2] - 1) {
            id = getIndexInputGrid(x, y, z, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x, y, z - 1, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            t2[2] = (double)flowMap[id + 2];

            du[2] = d_backwardDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[2] = d_backwardDiff(t1[1], t2[1], spacingSeedGrid[1]);
            dw[2] = d_backwardDiff(t1[2], t2[2], spacingSeedGrid[2]);
        } else {
            id = getIndexInputGrid(x, y, z - 1, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x, y, z + 1, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            t2[2] = (double)flowMap[id + 2];

            du[2] = d_centralDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[2] = d_centralDiff(t1[1], t2[1], spacingSeedGrid[1]);
            dw[2] = d_centralDiff(t1[2], t2[2], spacingSeedGrid[2]);

        }


        /// y Component
        if (y <= 0) {
            id = getIndexInputGrid(x, y, z, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x, y + 1, z, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            t2[2] = (double)flowMap[id + 2];

            du[1] = d_forwardDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[1] = d_forwardDiff(t1[1], t2[1], spacingSeedGrid[1]);
            dw[1] = d_forwardDiff(t1[2], t2[2], spacingSeedGrid[2]);

        } else if (y >= dimSeedGrid[1] - 1) {
            id = getIndexInputGrid(x, y, z, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x, y - 1, z, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            t2[2] = (double)flowMap[id + 2];

            du[1] = d_backwardDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[1] = d_backwardDiff(t1[1], t2[1], spacingSeedGrid[1]);
            dw[1] = d_backwardDiff(t1[2], t2[2], spacingSeedGrid[2]);

        } else {
            id = getIndexInputGrid(x, y - 1, z, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x, y + 1, z, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            t2[2] = (double)flowMap[id + 2];

            du[1] = d_centralDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[1] = d_centralDiff(t1[1], t2[1], spacingSeedGrid[1]);
            dw[1] = d_centralDiff(t1[2], t2[2], spacingSeedGrid[2]);
        }


        ///x Component
        if (x <= 0) {
            id = getIndexInputGrid(x, y, z, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x + 1, y, z, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            t2[2] = (double)flowMap[id + 2];

            du[0] = d_forwardDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[0] = d_forwardDiff(t1[1], t2[1], spacingSeedGrid[1]);
            dw[0] = d_forwardDiff(t1[2], t2[2], spacingSeedGrid[2]);
        } else if (x >= dimSeedGrid[0] - 1) {
            id = getIndexInputGrid(x, y, z, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x - 1, y, z, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            t2[2] = (double)flowMap[id + 2];

            du[0] = d_backwardDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[0] = d_backwardDiff(t1[1], t2[1], spacingSeedGrid[1]);
            dw[0] = d_backwardDiff(t1[2], t2[2], spacingSeedGrid[2]);


        } else {
            id = getIndexInputGrid(x - 1, y, z, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x + 1, y, z, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            t2[2] = (double)flowMap[id + 2];

            du[0] = d_centralDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[0] = d_centralDiff(t1[1], t2[1], spacingSeedGrid[1]);
            dw[0] = d_centralDiff(t1[2], t2[2], spacingSeedGrid[2]);

        }

        mat3 jacobi;
        d_mat3setrows(jacobi, du, dv, dw);
        mat3 jacobiT;
        mat3 cgTensor;
        d_mat3trp(jacobi, jacobiT);
        d_mat3mul(jacobiT, jacobi, cgTensor);
        double eMax = 0;
        vec3 eigenV;
        int realEigen = d_mat3eigenvalues(cgTensor, eigenV);
        eMax = fmax(fmax(eigenV[0], eigenV[1]), eigenV[2]);

        id = getIndexInputGrid(x, y, z, dimSeedGrid) / 3;
        if (realEigen != 3 || fabs(eMax) < d_LIMIT_DOUBLE) {
            // std::cerr<<"Eigenvalues of Cauchy Green Tensor are not real"<<endl;
            //cout<<realEigen<< " eigenvalues"<< eigenV[0]<< " " <<eigenV[1]<< " "<< eigenV[2]<<" "<<LIMIT_DOUBLE<<endl;
            eMax = 0;
            ftleField[id] = 0;
            return;
        }


        eMax = 1.0 / fabs(intTime) * log(sqrt(eMax));
        ftleField[id] = eMax;

    }
}
#endif
*/

////removing z components
#if !defined UNITTEST && !defined CPUEXEC
MAKEGLOBAL void computeFTLE(double* ftleField, double* flowMap, int* dimSeedGrid,
                            double* spacingSeedGrid, double intTime, double d_LIMIT_DOUBLE)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;


    ////int z = index / (dimSeedGrid[0] * dimSeedGrid[1]);
    int y = (index % (dimSeedGrid[0] * dimSeedGrid[1])) / dimSeedGrid[0];
    int x = (index % (dimSeedGrid[0] * dimSeedGrid[1])) % dimSeedGrid[0];
    //if(index < dimSeedGrid[0]*dimSeedGrid[1]*dimSeedGrid[2]) {
     //   int x = blockIdx.x * blockDim.x + threadIdx.x;//(index / (dimSeedGrid[0] * dimSeedGrid[1]));
     //   int y = blockIdx.y * blockDim.y + threadIdx.y;//(index % (dimSeedGrid[0] * dimSeedGrid[1])) / dimSeedGrid[1];
     //   int z = blockIdx.z * blockDim.z + threadIdx.z;//(index % (dimSeedGrid[0] * dimSeedGrid[1])) % dimSeedGrid[1];
    if (x < dimSeedGrid[0] && y < dimSeedGrid[1] &&
            x >=0 && y >=0){
        vec2 t1, t2;
        int id;
        vec2 du, dv, dw;

/*
        if (z <= 0) {
            id = getIndexInputGrid(x, y, z, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x, y, z + 1, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            t2[2] = (double)flowMap[id + 2];

            du[2] = d_forwardDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[2] = d_forwardDiff(t1[1], t2[1], spacingSeedGrid[1]);
            dw[2] = d_forwardDiff(t1[2], t2[2], spacingSeedGrid[2]);

        } else if (z >= dimSeedGrid[2] - 1) {
            id = getIndexInputGrid(x, y, z, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x, y, z - 1, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            t2[2] = (double)flowMap[id + 2];

            du[2] = d_backwardDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[2] = d_backwardDiff(t1[1], t2[1], spacingSeedGrid[1]);
            dw[2] = d_backwardDiff(t1[2], t2[2], spacingSeedGrid[2]);
        } else {
            id = getIndexInputGrid(x, y, z - 1, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x, y, z + 1, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            t2[2] = (double)flowMap[id + 2];

            du[2] = d_centralDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[2] = d_centralDiff(t1[1], t2[1], spacingSeedGrid[1]);
            dw[2] = d_centralDiff(t1[2], t2[2], spacingSeedGrid[2]);

        }
*/
	////removed z component
        /// y Component
        if (y <= 0) {
            id = getIndexInputGrid(x, y, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            ////t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x, y + 1, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            ////t2[2] = (double)flowMap[id + 2];

            du[1] = d_forwardDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[1] = d_forwardDiff(t1[1], t2[1], spacingSeedGrid[1]);
            ////dw[1] = d_forwardDiff(t1[2], t2[2], spacingSeedGrid[2]);

        } else if (y >= dimSeedGrid[1] - 1) {
            id = getIndexInputGrid(x, y, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            ////t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x, y - 1, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            ////t2[2] = (double)flowMap[id + 2];

            du[1] = d_backwardDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[1] = d_backwardDiff(t1[1], t2[1], spacingSeedGrid[1]);
            ////dw[1] = d_backwardDiff(t1[2], t2[2], spacingSeedGrid[2]);

        } else {
            id = getIndexInputGrid(x, y - 1, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            ////t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x, y + 1, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            ////t2[2] = (double)flowMap[id + 2];

            du[1] = d_centralDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[1] = d_centralDiff(t1[1], t2[1], spacingSeedGrid[1]);
            ////dw[1] = d_centralDiff(t1[2], t2[2], spacingSeedGrid[2]);
        }

	////removed z component
        ///x Component
        if (x <= 0) {
            id = getIndexInputGrid(x, y, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            ////t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x + 1, y, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            ////t2[2] = (double)flowMap[id + 2];

            du[0] = d_forwardDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[0] = d_forwardDiff(t1[1], t2[1], spacingSeedGrid[1]);
            ////dw[0] = d_forwardDiff(t1[2], t2[2], spacingSeedGrid[2]);
        } else if (x >= dimSeedGrid[0] - 1) {
            id = getIndexInputGrid(x, y, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            ////t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x - 1, y, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            ////t2[2] = (double)flowMap[id + 2];

            du[0] = d_backwardDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[0] = d_backwardDiff(t1[1], t2[1], spacingSeedGrid[1]);
            ////dw[0] = d_backwardDiff(t1[2], t2[2], spacingSeedGrid[2]);


        } else {
            id = getIndexInputGrid(x - 1, y, dimSeedGrid);
            t1[0] = (double)flowMap[id + 0];
            t1[1] = (double)flowMap[id + 1];
            ////t1[2] = (double)flowMap[id + 2];
            id = getIndexInputGrid(x + 1, y, dimSeedGrid);
            t2[0] = (double)flowMap[id + 0];
            t2[1] = (double)flowMap[id + 1];
            ////t2[2] = (double)flowMap[id + 2];

            du[0] = d_centralDiff(t1[0], t2[0], spacingSeedGrid[0]);
            dv[0] = d_centralDiff(t1[1], t2[1], spacingSeedGrid[1]);
            ////dw[0] = d_centralDiff(t1[2], t2[2], spacingSeedGrid[2]);

        }

        mat2 jacobi;
        d_mat2setrows(jacobi, du, dv);
        mat2 jacobiT;
        mat2 cgTensor;
        d_mat2trp(jacobi, jacobiT);
        d_mat2mul(jacobiT, jacobi, cgTensor);
        double eMax = 0;
        vec2 eigenV;
        int realEigen = d_mat2eigenvalues(cgTensor, eigenV);
	////changed fmax(fmax()) to fmax()
        eMax = fmax(eigenV[0], eigenV[1]);

        id = getIndexInputGrid(x, y, dimSeedGrid) / 2;
        if (realEigen != 2 || fabs(eMax) < d_LIMIT_DOUBLE) {
            // std::cerr<<"Eigenvalues of Cauchy Green Tensor are not real"<<endl;
            //cout<<realEigen<< " eigenvalues"<< eigenV[0]<< " " <<eigenV[1]<< " "<< eigenV[2]<<" "<<LIMIT_DOUBLE<<endl;
            eMax = 0;
            ftleField[id] = 0;
            return;
        }


        eMax = 1.0 / fabs(intTime) * log(sqrt(eMax));
        ftleField[id] = eMax;

    }
}
#endif

/*
MAKEGLOBAL void
calcFtleField(float *inputArray, double*flowMap, double *ftleField, int *dimInputGrid, int *dimSeedGrid,
              double *spacingInputGrid,
              double *spacingSeedGrid, double *origin, double* originSource, int integrationDirection, double stepSize, double intTime,
              double stagnationThresh, double d_LIMIT_DOUBLE, int maxNumberOfSteps, int CPUindex = 0) {
    //int index = getGlobalIdx();
    ///Calc global thresh index
    int index;
#ifdef CPUEXEC
    index = CPUindex;
#else
    index = blockDim.x * blockIdx.x + threadIdx.x;
#endif

    int zIndex = index / (dimSeedGrid[0] * dimSeedGrid[1]);
    //blockIdx.x * blockDim.x + threadIdx.x;//(index / (dimSeedGrid[0] * dimSeedGrid[1]));
    int yIndex = (index % (dimSeedGrid[0] * dimSeedGrid[1])) / dimSeedGrid[0];
    //blockIdx.y * blockDim.y + threadIdx.y;//(index % (dimSeedGrid[0] * dimSeedGrid[1])) / dimSeedGrid[1];
    int xIndex = (index % (dimSeedGrid[0] * dimSeedGrid[1])) % dimSeedGrid[0];
    //blockIdx.z * blockDim.z + threadIdx.z;//(index % (dimSeedGrid[0] * dimSeedGrid[1])) % dimSeedGrid[1];


    if (xIndex < dimSeedGrid[0] && yIndex < dimSeedGrid[1] && zIndex < dimSeedGrid[2]&&
            xIndex >=0 && yIndex >=0 && zIndex|>=0)

    {
        vec3 location;
        double xLocation = double(xIndex) *  spacingSeedGrid[0] + originSource[0];//blockIdx.x * blockDim.x + threadIdx.x * spacingSeedGrid[0];
        double yLocation = double(yIndex) *  spacingSeedGrid[1] + originSource[1];//blockIdx.y * blockDim.y + threadIdx.y * spacingSeedGrid[1];
        double zLocation = double(zIndex) *  spacingSeedGrid[2] + originSource[2];//blockIdx.z * blockDim.z + threadIdx.z * spacingSeedGrid[2];


        d_vec3set(location,xLocation,yLocation,zLocation);

        double currentIntTime = 0.;
        double currentStagnation = 0;
        int maxSteps = maxNumberOfSteps;
        int currentSteps = 0;

        /// Integrate Streamline


        while (currentIntTime <= intTime) {
            currentIntTime += integratePoint(location, inputArray, origin, dimInputGrid, spacingInputGrid,
                                             integrationDirection, stepSize, currentStagnation);
            ///Check for stagnation of integration
            if (currentStagnation <= stagnationThresh)
                break;
            ++currentSteps;
            if(currentSteps >= maxSteps)
                break;
        }
        ///Set flow map (x,y,z)
        flowMap[3 * index] = location[0];
        flowMap[3 * index + 1] = location[1];
        flowMap[3 * index + 2] = location[2];
        //ftleField[index] = d_vec3mag(location);
        //TODO add finite differences when sync issue is fixed
        //computeFTLE(ftleField, flowMap , dimSeedGrid , spacingSeedGrid , intTime, d_LIMIT_DOUBLE);
        //d_vec3print(location, " location: ");
        //printf("SeedGrid: %d %d %d \n", dimSeedGrid[0],dimSeedGrid[1],dimSeedGrid[2]);
        //printf("Index: %d %d %d %d %f \n",xIndex, yIndex,zIndex, index, ftleField[index]);





    }

    return;
}
*/

MAKEGLOBAL void
calcFtleField(float *inputArray, double*flowMap, double *ftleField, int *dimInputGrid, int *dimSeedGrid,
              double *spacingInputGrid,
              double *spacingSeedGrid, double *origin, double* originSource, int integrationDirection, double stepSize, double intTime,
              double stagnationThresh, double d_LIMIT_DOUBLE, int maxNumberOfSteps, int CPUindex = 0) {
    //int index = getGlobalIdx();
    ///Calc global thresh index
    int index;
#ifdef CPUEXEC
    index = CPUindex;
#else
    index = blockDim.x * blockIdx.x + threadIdx.x;
#endif

////removed z component
    ////int zIndex = index / (dimSeedGrid[0] * dimSeedGrid[1]);
    //blockIdx.x * blockDim.x + threadIdx.x;//(index / (dimSeedGrid[0] * dimSeedGrid[1]));
    int yIndex = (index % (dimSeedGrid[0] * dimSeedGrid[1])) / dimSeedGrid[0];
    //blockIdx.y * blockDim.y + threadIdx.y;//(index % (dimSeedGrid[0] * dimSeedGrid[1])) / dimSeedGrid[1];
    int xIndex = (index % (dimSeedGrid[0] * dimSeedGrid[1])) % dimSeedGrid[0];
    //blockIdx.z * blockDim.z + threadIdx.z;//(index % (dimSeedGrid[0] * dimSeedGrid[1])) % dimSeedGrid[1];

////removed z component
    if (xIndex < dimSeedGrid[0] && yIndex < dimSeedGrid[1] &&
            xIndex >=0 && yIndex >=0)


    {
        vec2 location;
        double xLocation = double(xIndex) *  spacingSeedGrid[0] + originSource[0];//blockIdx.x * blockDim.x + threadIdx.x * spacingSeedGrid[0];
        double yLocation = double(yIndex) *  spacingSeedGrid[1] + originSource[1];//blockIdx.y * blockDim.y + threadIdx.y * spacingSeedGrid[1];
        ////double zLocation = double(zIndex) *  spacingSeedGrid[2] + originSource[2];//blockIdx.z * blockDim.z + threadIdx.z * spacingSeedGrid[2];


        d_vec2set(location,xLocation,yLocation);

        double currentIntTime = 0.;
        double currentStagnation = 0;
        int maxSteps = maxNumberOfSteps;
        int currentSteps = 0;

        /// Integrate Streamline


        while (currentIntTime <= intTime) {
            currentIntTime += integratePoint(location, inputArray, origin, dimInputGrid, spacingInputGrid,
                                             integrationDirection, stepSize, currentStagnation);
            ///Check for stagnation of integration
            if (currentStagnation <= stagnationThresh)
                break;
            ++currentSteps;
            if(currentSteps >= maxSteps)
                break;
        }

////changed 3 to 2
        ///Set flow map (x,y)
        flowMap[2 * index] = location[0];
        flowMap[2 * index + 1] = location[1];
        ////flowMap[3 * index + 2] = location[2];
        //ftleField[index] = d_vec3mag(location);
        //TODO add finite differences when sync issue is fixed
        //computeFTLE(ftleField, flowMap , dimSeedGrid , spacingSeedGrid , intTime, d_LIMIT_DOUBLE);
        //d_vec3print(location, " location: ");
        //printf("SeedGrid: %d %d %d \n", dimSeedGrid[0],dimSeedGrid[1],dimSeedGrid[2]);
        //printf("Index: %d %d %d %d %f \n",xIndex, yIndex,zIndex, index, ftleField[index]);





    }

    return;
}



CudaIntegrator::~CudaIntegrator() {}

/*
void CudaIntegrator::
inInterpolator(vec3 a, vec3 b, vec3 c, vec3 d, vec3 e, vec3 f, vec3 g, vec3 h, double spacingX,
                                        double spacingY, double spacingZ, vec3 location, vec3 &output) {


}
*/
////removed component e,f,g,h, spacingZ and changed 3 to 2 and changed name to bilin
void CudaIntegrator::bilinInterpolator(vec2 a, vec2 b, vec2 c, vec2 d, double spacingX,
                                        double spacingY, vec2 location, vec2 &output) {


}

/*
double *CudaIntegrator::integrate(float *inputArray, double* flowMap) {
    //double* flowMap;
    double *d_flowMap;
    double *ftleField;
    double *d_ftleField;
    float *d_inputArray;
    int *d_dimSeedGrid;
    int *d_dimInputGrid;
    double *d_spacingSeedGrid;
    double *d_spacingInputGrid;
    double *d_origin;
    double *d_originSource;
    int sizeFlowMap = sizeof(double) * this->numPointsSeedGrid * 3;
    int sizeFtleField = sizeof(double) * this->numPointsSeedGrid;
    int sizeInputArray = sizeof(float) * this->numPointsInputGrid * 3;
    int sizeDim = sizeof(int) * 3;
    int sizeSpacing = sizeof(double) * 3;


    ///Alloc host and device memory
    //flowMap = (double*) malloc(sizeFlowMap);
    ftleField = (double *) malloc(sizeFtleField);
*/

////changed 3 to 2
double *CudaIntegrator::integrate(float *inputArray, double* flowMap) {
    //double* flowMap;
    double *d_flowMap;
    double *ftleField;
    double *d_ftleField;
    float *d_inputArray;
    int *d_dimSeedGrid;
    int *d_dimInputGrid;
    double *d_spacingSeedGrid;
    double *d_spacingInputGrid;
    double *d_origin;
    double *d_originSource;
    int sizeFlowMap = sizeof(double) * this->numPointsSeedGrid * 2;
    int sizeFtleField = sizeof(double) * this->numPointsSeedGrid;
    int sizeInputArray = sizeof(float) * this->numPointsInputGrid * 2;
    int sizeDim = sizeof(int) * 2;
    int sizeSpacing = sizeof(double) * 2;


    ///Alloc host and device memory
    //flowMap = (double*) malloc(sizeFlowMap);
    ftleField = (double *) malloc(sizeFtleField);

/*
#ifndef UNITTEST
    hipMalloc((void **) &d_flowMap, sizeFlowMap);
    hipMalloc((void **) &d_ftleField, sizeFtleField);
    hipMalloc((void **) &d_inputArray, sizeInputArray);
    hipMalloc((void **) &d_dimSeedGrid, sizeDim);
    hipMalloc((void **) &d_dimInputGrid, sizeDim);
    hipMalloc((void **) &d_spacingSeedGrid, sizeSpacing);
    hipMalloc((void **) &d_spacingInputGrid, sizeSpacing);
    hipMalloc((void **) &d_origin, sizeSpacing);
    hipMalloc((void **) &d_originSource, sizeSpacing);

    ///Set sizes for block and threads
    dim3 gridDim = dim3(this->dimensionsSeedGrid[0] /8 +1 , this->dimensionsSeedGrid[1] / 8 +1 , this->dimensionsSeedGrid[2] /8 +1);
    dim3 blockDim = dim3(8, 8, 8);

    int M = this->dimensionsSeedGrid[0] * this->dimensionsSeedGrid[1] *
            this->dimensionsSeedGrid[2];
    int N = 256;

    //for(int i=0 ; i<numPointsSeedGrid; ++i)
      //  printf("Location.x %i: %i \n",i ,ftleField[i]);

#endif
*/

#ifndef UNITTEST
    hipMalloc((void **) &d_flowMap, sizeFlowMap);
    hipMalloc((void **) &d_ftleField, sizeFtleField);
    hipMalloc((void **) &d_inputArray, sizeInputArray);
    hipMalloc((void **) &d_dimSeedGrid, sizeDim);
    hipMalloc((void **) &d_dimInputGrid, sizeDim);
    hipMalloc((void **) &d_spacingSeedGrid, sizeSpacing);
    hipMalloc((void **) &d_spacingInputGrid, sizeSpacing);
    hipMalloc((void **) &d_origin, sizeSpacing);
    hipMalloc((void **) &d_originSource, sizeSpacing);

////changed 3 to 2
    ///Set sizes for block and threads
    dim3 gridDim = dim3(this->dimensionsSeedGrid[0] /8 +1 , this->dimensionsSeedGrid[1] /8 +1);
    dim3 blockDim = dim3(8, 8);

    ////changed 256 to 32
    int M = this->dimensionsSeedGrid[0] * this->dimensionsSeedGrid[1];
    int N = 32;

    //for(int i=0 ; i<numPointsSeedGrid; ++i)
      //  printf("Location.x %i: %i \n",i ,ftleField[i]);

#endif



#ifndef CPUEXEC
    ///Copy data to device
    printf("Start GPU Kernel\n");
    hipMemcpy(d_flowMap, flowMap, sizeFlowMap, hipMemcpyHostToDevice);
    hipMemcpy(d_ftleField, ftleField, sizeFtleField, hipMemcpyHostToDevice);
    hipMemcpy(d_inputArray, inputArray, sizeInputArray, hipMemcpyHostToDevice);
    hipMemcpy(d_dimSeedGrid, this->dimensionsSeedGrid, sizeDim, hipMemcpyHostToDevice);
    hipMemcpy(d_dimInputGrid, this->dimensionsInputGrid, sizeDim, hipMemcpyHostToDevice);
    hipMemcpy(d_spacingSeedGrid, this->spacingSeedGrid, sizeSpacing, hipMemcpyHostToDevice);
    hipMemcpy(d_spacingInputGrid, this->spacingInputGrid, sizeSpacing, hipMemcpyHostToDevice);
    hipMemcpy(d_origin, this->origin, sizeSpacing, hipMemcpyHostToDevice);
    hipMemcpy(d_originSource, this->originSource, sizeSpacing, hipMemcpyHostToDevice);




    calcFtleField <<< M / N +1, N >>>
                                     (d_inputArray, d_flowMap, d_ftleField, d_dimInputGrid, d_dimSeedGrid,
                                             d_spacingInputGrid, d_spacingSeedGrid,
                                             d_origin, d_originSource, this->integrationDirection,
                                             this->stepSize, this->integrationTime,
                                             this->stagnationThreshold, LIMIT_DOUBLE, this->maxNumberOfSteps);


    hipDeviceSynchronize();

    computeFTLE<<< M / N +1, N >>>
            (d_ftleField, d_flowMap , d_dimSeedGrid , d_spacingSeedGrid , this->integrationTime, LIMIT_DOUBLE);

    ///Copy data from device to host
    hipDeviceSynchronize();
    hipMemcpy(ftleField, d_ftleField, sizeFtleField, hipMemcpyDeviceToHost);
    hipMemcpy(flowMap, d_flowMap, sizeFlowMap, hipMemcpyDeviceToHost);



    //for(int i=0 ; i<numPointsSeedGrid; ++i)
      //  printf("Location.x %i: %f \n",i ,ftleField[i]);


    hipError_t error = hipGetLastError();
    if(error!=hipSuccess)
    {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        exit(-1);
    }
    //free(flowMap);
    //free(ftleField);
    hipFree(d_flowMap);
    hipFree(d_ftleField);
    hipFree(d_inputArray);
    hipFree(d_dimSeedGrid);
    hipFree(d_dimInputGrid);
    hipFree(d_spacingSeedGrid);
    hipFree(d_spacingInputGrid);
    hipFree(d_origin);
#else
    for(int i = 0; i < this->numPointsSeedGrid; ++i) {
        calcFtleField(inputArray, flowMap, ftleField, dimensionsInputGrid, dimensionsSeedGrid,
                      spacingInputGrid, spacingSeedGrid,
                      origin, originSource, this->integrationDirection,
                      this->stepSize, this->integrationTime,
                      this->stagnationThreshold, LIMIT_DOUBLE, this->maxNumberOfSteps, i);


       // computeFTLE(d_ftleField, d_flowMap, d_dimSeedGrid, d_spacingSeedGrid, this->integrationTime, LIMIT_DOUBLE);
    }
#endif

    return ftleField;

}

